/***********************************************************************
  Copyright (C) 2020 Hironori Fujimoto

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************/
#include "flightlib/sensors/sgm_gpu/sgm_gpu.h"

#include "flightlib/sensors/sgm_gpu/cost_aggregation.h"
#include "flightlib/sensors/sgm_gpu/costs.h"
#include "flightlib/sensors/sgm_gpu/hamming_cost.h"
#include "flightlib/sensors/sgm_gpu/left_right_consistency.h"
#include "flightlib/sensors/sgm_gpu/median_filter.h"

namespace sgm_gpu {
// Variables which have CUDA-related type are put here
//   in order to include sgm_gpu.h from non-CUDA package
hipStream_t stream1_;
hipStream_t stream2_;
hipStream_t stream3_;

dim3 BLOCK_SIZE_;
dim3 grid_size_;

SgmGpu::SgmGpu(const int cols, const int rows)
    : memory_allocated_(false), cols_(cols), rows_(rows) {
  // Get parameters used in SGM algorithm
  p1_ = 6;   // static_cast<uint8_t>(private_node_handle_->param("p1", 6));
  p2_ = 96;  // static_cast<uint8_t>(private_node_handle_->param("p2", 96));
  check_consistency_ = true;  // private_node_handle_->param("check_consistency", true);

  // Create streams
  hipStreamCreate(&stream1_);
  hipStreamCreate(&stream2_);
  hipStreamCreate(&stream3_);
}

SgmGpu::~SgmGpu() {
  freeMemory();

  // 设置为 nullptr，防止重复销毁
  if (stream1_) {
	  hipStreamDestroy(stream1_);
	  stream1_ = nullptr;
  }
  if (stream2_) {
	  hipStreamDestroy(stream2_);
	  stream2_ = nullptr;
  }
  if (stream3_) {
	  hipStreamDestroy(stream3_);
	  stream3_ = nullptr;
  }
}

void SgmGpu::allocateMemory(uint32_t cols, uint32_t rows) {
  freeMemory();

  cols_ = cols;
  rows_ = rows;

  int total_pixel = cols_ * rows_;
  hipMalloc((void **)&d_im0_, sizeof(uint8_t) * total_pixel);
  hipMalloc((void **)&d_im1_, sizeof(uint8_t) * total_pixel);

  hipMalloc((void **)&d_transform0_, sizeof(cost_t) * total_pixel);
  hipMalloc((void **)&d_transform1_, sizeof(cost_t) * total_pixel);

  int cost_volume_size = total_pixel * MAX_DISPARITY;
  hipMalloc((void **)&d_cost_, sizeof(uint8_t) * cost_volume_size);

  hipMalloc((void **)&d_L0_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L1_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L2_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L3_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L4_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L5_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L6_, sizeof(uint8_t) * cost_volume_size);
  hipMalloc((void **)&d_L7_, sizeof(uint8_t) * cost_volume_size);

  hipMalloc((void **)&d_s_, sizeof(uint16_t) * cost_volume_size);

  hipMalloc((void **)&d_disparity_, sizeof(uint8_t) * total_pixel);
  hipMalloc((void **)&d_disparity_filtered_uchar_,
             sizeof(uint8_t) * total_pixel);
  hipMalloc((void **)&d_disparity_right_, sizeof(uint8_t) * total_pixel);
  hipMalloc((void **)&d_disparity_right_filtered_uchar_,
             sizeof(uint8_t) * total_pixel);

  memory_allocated_ = true;
}

void SgmGpu::freeMemory() {
  if (!memory_allocated_) return;

  hipFree(d_im0_);
  hipFree(d_im1_);
  hipFree(d_transform0_);
  hipFree(d_transform1_);
  hipFree(d_L0_);
  hipFree(d_L1_);
  hipFree(d_L2_);
  hipFree(d_L3_);
  hipFree(d_L4_);
  hipFree(d_L5_);
  hipFree(d_L6_);
  hipFree(d_L7_);
  hipFree(d_disparity_);
  hipFree(d_disparity_filtered_uchar_);
  hipFree(d_disparity_right_);
  hipFree(d_disparity_right_filtered_uchar_);
  hipFree(d_cost_);
  hipFree(d_s_);

  memory_allocated_ = false;
}

bool SgmGpu::computeDisparity(const cv::Mat &left_image,
                              const cv::Mat &right_image,
                              cv::Mat &disparity_out) {
  // Convert images to grayscale
  cv::Mat left_mono8, right_mono8;
  if (left_image.channels() > 1) {
    cv::cvtColor(left_image, left_mono8, CV_RGB2GRAY);
  }

  if (right_image.channels() > 1) {
    cv::cvtColor(right_image, right_mono8, CV_RGB2GRAY);
  }

  // Resize images to their width and height divisible by 4 for limit of CUDA
  // code
  resizeToDivisibleBy4(left_mono8, right_mono8);

  // Reallocate memory if needed
  bool size_changed = (cols_ != left_mono8.cols || rows_ != left_mono8.rows);
  if (!memory_allocated_ || size_changed)
    allocateMemory(left_mono8.cols, left_mono8.rows);

  // Copy image to GPU device
  size_t mono8_image_size = left_mono8.total() * sizeof(uint8_t);
  hipMemcpyAsync(d_im0_, left_mono8.ptr<uint8_t>(), mono8_image_size,
                  hipMemcpyHostToDevice, stream1_);
  hipMemcpyAsync(d_im1_, right_mono8.ptr<uint8_t>(), mono8_image_size,
                  hipMemcpyHostToDevice, stream1_);

  BLOCK_SIZE_.x = 32;
  BLOCK_SIZE_.y = 32;

  grid_size_.x = (cols_ + BLOCK_SIZE_.x - 1) / BLOCK_SIZE_.x;
  grid_size_.y = (rows_ + BLOCK_SIZE_.y - 1) / BLOCK_SIZE_.y;

  CenterSymmetricCensusKernelSM2<<<grid_size_, BLOCK_SIZE_, 0, stream1_>>>(
      d_im0_, d_im1_, d_transform0_, d_transform1_, rows_, cols_);

  hipStreamSynchronize(stream1_);
  HammingDistanceCostKernel<<<rows_, MAX_DISPARITY, 0, stream1_>>>(
      d_transform0_, d_transform1_, d_cost_, rows_, cols_);

  const int PIXELS_PER_BLOCK = COSTAGG_BLOCKSIZE / WARP_SIZE;
  const int PIXELS_PER_BLOCK_HORIZ = COSTAGG_BLOCKSIZE_HORIZ / WARP_SIZE;

  // Cost Aggregation
  CostAggregationKernelLeftToRight<<<(rows_ + PIXELS_PER_BLOCK_HORIZ - 1) /
                                         PIXELS_PER_BLOCK_HORIZ,
                                     COSTAGG_BLOCKSIZE_HORIZ, 0, stream2_>>>(
      d_cost_, d_L0_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
      d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_, d_L4_, d_L5_,
      d_L6_);
  CostAggregationKernelRightToLeft<<<(rows_ + PIXELS_PER_BLOCK_HORIZ - 1) /
                                         PIXELS_PER_BLOCK_HORIZ,
                                     COSTAGG_BLOCKSIZE_HORIZ, 0, stream3_>>>(
      d_cost_, d_L1_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
      d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_, d_L4_, d_L5_,
      d_L6_);
  CostAggregationKernelUpToDown<<<(cols_ + PIXELS_PER_BLOCK - 1) /
                                      PIXELS_PER_BLOCK,
                                  COSTAGG_BLOCKSIZE, 0, stream1_>>>(
      d_cost_, d_L2_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
      d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_, d_L4_, d_L5_,
      d_L6_);
  CostAggregationKernelDownToUp<<<(cols_ + PIXELS_PER_BLOCK - 1) /
                                      PIXELS_PER_BLOCK,
                                  COSTAGG_BLOCKSIZE, 0, stream1_>>>(
      d_cost_, d_L3_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
      d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_, d_L4_, d_L5_,
      d_L6_);
  CostAggregationKernelDiagonalDownUpLeftRight<<<
      (cols_ + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0,
      stream1_>>>(d_cost_, d_L4_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
                  d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_,
                  d_L4_, d_L5_, d_L6_);
  CostAggregationKernelDiagonalUpDownLeftRight<<<
      (cols_ + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0,
      stream1_>>>(d_cost_, d_L5_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
                  d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_,
                  d_L4_, d_L5_, d_L6_);
  CostAggregationKernelDiagonalDownUpRightLeft<<<
      (cols_ + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0,
      stream1_>>>(d_cost_, d_L6_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
                  d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_,
                  d_L4_, d_L5_, d_L6_);
  CostAggregationKernelDiagonalUpDownRightLeft<<<
      (cols_ + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0,
      stream1_>>>(d_cost_, d_L7_, d_s_, p1_, p2_, rows_, cols_, d_transform0_,
                  d_transform1_, d_disparity_, d_L0_, d_L1_, d_L2_, d_L3_,
                  d_L4_, d_L5_, d_L6_);

  int total_pixel = rows_ * cols_;
  MedianFilter3x3<<<(total_pixel + MAX_DISPARITY - 1) / MAX_DISPARITY,
                    MAX_DISPARITY, 0, stream1_>>>(
      d_disparity_, d_disparity_filtered_uchar_, rows_, cols_);

  if (check_consistency_) {
    ChooseRightDisparity<<<grid_size_, BLOCK_SIZE_, 0, stream1_>>>(
        d_disparity_right_, d_s_, rows_, cols_);
    MedianFilter3x3<<<(total_pixel + MAX_DISPARITY - 1) / MAX_DISPARITY,
                      MAX_DISPARITY, 0, stream1_>>>(
        d_disparity_right_, d_disparity_right_filtered_uchar_, rows_, cols_);

    LeftRightConsistencyCheck<<<grid_size_, BLOCK_SIZE_, 0, stream1_>>>(
        d_disparity_filtered_uchar_, d_disparity_right_filtered_uchar_, rows_,
        cols_);
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("libsgm_gpu ERROR: %s %d\n", hipGetErrorString(err), err);
    return false;
  }

  hipDeviceSynchronize();
  cv::Mat disparity(rows_, cols_, CV_8UC1);
  hipMemcpy(disparity.data, d_disparity_filtered_uchar_,
             sizeof(uint8_t) * total_pixel, hipMemcpyDeviceToHost);

  // Restore image size if resized to be divisible by 4
  if (cols_ != left_image.cols || rows_ != left_image.rows) {
    cv::Size input_size(left_image.cols, left_image.rows);
    cv::resize(disparity, disparity, input_size, 0, 0, cv::INTER_AREA);
  }

  disparity_out = disparity;
  //  convertToMsg(disparity, left_camera_info, right_camera_info,
  //  disparity_msg);

  return true;
}

void SgmGpu::resizeToDivisibleBy4(cv::Mat &left_image, cv::Mat &right_image) {
  bool need_resize = false;
  cv::Size original_size, resized_size;

  original_size = cv::Size(left_image.cols, left_image.rows);
  resized_size = original_size;
  if (original_size.width % 4 != 0) {
    need_resize = true;
    resized_size.width = (original_size.width / 4 + 1) * 4;
  }
  if (original_size.height % 4 != 0) {
    need_resize = true;
    resized_size.height = (original_size.height / 4 + 1) * 4;
  }

  if (need_resize) {
    cv::resize(left_image, left_image, resized_size, 0, 0, cv::INTER_LINEAR);
    cv::resize(right_image, right_image, resized_size, 0, 0, cv::INTER_LINEAR);
  }
}


}  // namespace sgm_gpu
